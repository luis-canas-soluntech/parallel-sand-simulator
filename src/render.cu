#include "hip/hip_runtime.h"
#include <SDL2/SDL.h>
#include "/usr/include/SDL2/SDL_ttf.h"
#include <stdbool.h> 
#include <string.h>
#include <iostream>
#include <stdlib.h>
#include <sys/time.h>
#include <ctime>
#include "hip/hip_runtime.h"
#include "common/book.h"

#include "logic.h"
#include "render.h"
#include "util.h"


int mod(int a, int b)
{
    int r = a % b;
    return r < 0 ? r + b : r;
}

//!! define color of the automate, already listed in logic properties

const SDL_Color BLACK_CELL_COLOR = { .r = 0, .g = 0, .b = 0 };
const SDL_Color WHITE_CELL_COLOR = { .r = 255, .g = 255, .b = 255 };
const SDL_Color BLUE_CELL_COLOR = { .r = 0, .g = 0, .b = 255 };
const SDL_Color RED_CELL_COLOR = { .r = 255, .g = 0, .b = 0 };
const SDL_Color GRAY_CELL_COLOR = { .r = 128, .g = 128, .b = 128 };
const SDL_Color YELLOW_CELL_COLOR = { .r = 255, .g = 255, .b = 0 };
const SDL_Color PURPLE_CELL_COLOR = { .r = 119, .g = 0, .b = 199 };
const SDL_Color GREEN_CELL_COLOR = { .r = 0, .g = 255, .b = 0 };
const SDL_Color WHITEBLUE_CELL_COLOR = { .r = 176, .g = 241, .b = 247 };
const SDL_Color GRAYSMOKE_CELL_COLOR = { .r = 176, .g = 176, .b = 176 };
const SDL_Color STRUCTURE_CELL_COLOR = { .r = 67, .g = 59, .b = 27 };
const SDL_Color ANT_COLOR = { .r = 255, .g = 50, .b = 50 };

//NUMBER OF THREADS WE ARE USING IN SPECIFIC MOMENT
int threads=1;
int s_cambio_threads = 10; //Segundos para aumentar las thread

//Variables benchmarking Render Grid
int render_grid_framecnt = 0;
int fps_render_grid = 0;
double avg_time_render_grid = 0;


void render_grid(SDL_Renderer *renderer, const state_t *state)
{
    // //*calculate time to render the grid
    // struct timeval tval_before, tval_after, tval_result;
    
    // gettimeofday(&tval_before, NULL);


    //* Change thread number every 5 seconds and check if threads are less than the maximum of threads, so we can see the FPS for each number of threads
    ++render_grid_framecnt; //Ir sumando los frames

    resetTimer(TVAL_THREAD_2); //Actualizar timer
    
    if(( getTimerS(TVAL_THREAD_2)-getTimerS(TVAL_THREAD_1) >= s_cambio_threads) && (threads <= THREADS)){
        
        double avg_fps = render_grid_framecnt/s_cambio_threads;
        long double avg_time = ((avg_time_render_grid/avg_fps)/s_cambio_threads)*1000000;
        fps_render_grid = avg_fps;
        SDL_Log("[RENDER] Threads: %d, #de FPS promedio de los anteriores %d segundos: %0.1f, Tiempo promedio (ms): %0.1Lf", 
        threads, 
        s_cambio_threads, 
        avg_fps,
        avg_time);
        threads ++; //aumenta # threads
        resetTimer(TVAL_THREAD_2); //Actualizar timer
        resetTimer(TVAL_THREAD_1); //Actualizar timer
        render_grid_framecnt = 0; //Reiniciar la cuenta
        avg_time_render_grid = 0; //Reniciar el promedio de tiempo en X segundos
       
    }

    
    //#pragma omp parallel num_threads(threads)  
    //{
    //#pragma omp for
    for (int x = 0; x < N; x++){  
        for (int y = 0; y < N; y++) {
              SDL_Rect rect = {
                  .x = x * CELL_WIDTH,
                  .y = y * CELL_HEIGHT,
                  .w = CELL_WIDTH,
                  .h = CELL_HEIGHT
              };

              switch(state->board[x][y]) {
                  case BLACK:
                      SDL_SetRenderDrawColor(renderer, BLACK_CELL_COLOR.r, BLACK_CELL_COLOR.g, BLACK_CELL_COLOR.b, 255);
                      SDL_RenderFillRect(renderer, &rect);
                      break;

                  case BLUE:
                      SDL_SetRenderDrawColor(renderer, BLUE_CELL_COLOR.r, BLUE_CELL_COLOR.g, BLUE_CELL_COLOR.b, 255);
                      SDL_RenderFillRect(renderer, &rect);
                      break;

                  case RED:
                      SDL_SetRenderDrawColor(renderer, RED_CELL_COLOR.r, RED_CELL_COLOR.g, RED_CELL_COLOR.b, 255);
                      SDL_RenderFillRect(renderer, &rect);
                      break;
                  case GRAY:
                      SDL_SetRenderDrawColor(renderer, GRAY_CELL_COLOR.r, GRAY_CELL_COLOR.g, GRAY_CELL_COLOR.b, 255);
                      SDL_RenderFillRect(renderer, &rect);
                      break;
                  case YELLOW:
                      SDL_SetRenderDrawColor(renderer, YELLOW_CELL_COLOR.r, YELLOW_CELL_COLOR.g, YELLOW_CELL_COLOR.b, 255);
                      SDL_RenderFillRect(renderer, &rect);
                      break;
                  case WHITEBLUE:
                      SDL_SetRenderDrawColor(renderer, WHITEBLUE_CELL_COLOR.r, WHITEBLUE_CELL_COLOR.g, WHITEBLUE_CELL_COLOR.b, 255);
                      SDL_RenderFillRect(renderer, &rect);
                      break;
                  case GREEN:
                      SDL_SetRenderDrawColor(renderer, GREEN_CELL_COLOR.r, GREEN_CELL_COLOR.g, GREEN_CELL_COLOR.b, 255);
                      SDL_RenderFillRect(renderer, &rect);
                      break;
                    
                  case PURPLE:
                      SDL_SetRenderDrawColor(renderer, PURPLE_CELL_COLOR.r, PURPLE_CELL_COLOR.g, PURPLE_CELL_COLOR.b, 255);
                      SDL_RenderFillRect(renderer, &rect);
                      break;
                  case GRAYSMOKE:
                      SDL_SetRenderDrawColor(renderer, GRAYSMOKE_CELL_COLOR.r, GRAYSMOKE_CELL_COLOR.g, GRAYSMOKE_CELL_COLOR.b, 255);
                      SDL_RenderFillRect(renderer, &rect);
                      break;
                  case STRUCTURE:
                      SDL_SetRenderDrawColor(renderer, STRUCTURE_CELL_COLOR.r, STRUCTURE_CELL_COLOR.g, STRUCTURE_CELL_COLOR.b, 255);
                      SDL_RenderFillRect(renderer, &rect);
                      break;
                  default: {}
              }
          }}


        // //*calculate time to render the grid
        // gettimeofday(&tval_after, NULL);
        // timersub(&tval_after, &tval_before, &tval_result);
        


        //* calculate total TIME to run the whole program
        resetTimer(TVAL_TOTAL_2);

        long double  d = ((getTimerS(TVAL_TOTAL_2)*1000000+(getTimerMS(TVAL_TOTAL_2))) -(getTimerS(TVAL_TOTAL_1)*1000000+(getTimerMS(TVAL_TOTAL_1)) ));

        avg_time_render_grid += d/1000000;

        char str[128];
        sprintf(str, "Total time to loop the whole program (ms): %0.1Lf", 
            d
           );
        renderFormattedText(renderer, str, 0 , 20);

        char str2[128];
        sprintf(str2, "Threads: %d, AVG_FPS(%d s): %d", threads, s_cambio_threads, fps_render_grid);
        renderFormattedText(renderer, str2, 250 , 0);

        //* calculate total time to run the whole program
        resetTimer(TVAL_TOTAL_1); 
    //}

}

void langtons_ant(SDL_Renderer *renderer, state_t *state)
{
    // RENDER ANT
    SDL_SetRenderDrawColor(renderer, ANT_COLOR.r, ANT_COLOR.g, ANT_COLOR.b, 255);
    SDL_Rect ant_rect = {
        .x = state->ant.x * CELL_WIDTH ,
        .y = state->ant.y * CELL_HEIGHT,
        .w = CELL_WIDTH,
        .h = CELL_HEIGHT
    };
    SDL_RenderFillRect(renderer, &ant_rect);

    if (state->mode == RUNNING_MODE)
    for (int i = 0; i < MOVES_PER_FRAME; i++) {
        int current = state->board[state->ant.x][state->ant.y];

        // TURN 90º
        switch (current) {
            case WHITE:
                state->ant.dir = mod(state->ant.dir + 1, 4);
                break;
            case BLACK:
                state->ant.dir = mod(state->ant.dir - 1, 4);
                break;
        }

        // FLIP THE COLOR OF THE SQUARE
        state->board[state->ant.x][state->ant.y] = BLACK + WHITE - current;

        // MOVE FORWARD ONE UNIT
        switch (state->ant.dir) {
            case UP:
                state->ant.y = mod(state->ant.y - 1, N);
                break;
            case RIGHT:
                state->ant.x = mod(state->ant.x + 1, N);
                break;
            case DOWN:
                state->ant.y = mod(state->ant.y + 1, N);
                break;
            case LEFT:
                state->ant.x = mod(state->ant.x - 1, N);
                break;
        }
    }
}

void game_of_life(SDL_Renderer *renderer, state_t *state)
{
    if (state->mode == RUNNING_MODE)
    for (int i = 0; i < MOVES_PER_FRAME; i++) {
        int new_board[N][N];

        for (int x = 0; x < N; x++)
            for (int y = 0; y < N; y++) {
                int n_neigh = 
                    state->board[mod((x - 1), N)][mod((y - 1), N)] +
                    state->board[mod((x    ), N)][mod((y - 1), N)] +
                    state->board[mod((x + 1), N)][mod((y - 1), N)] +
                    state->board[mod((x - 1), N)][mod((y    ), N)] +
                    state->board[mod((x + 1), N)][mod((y    ), N)] +
                    state->board[mod((x - 1), N)][mod((y + 1), N)] +
                    state->board[mod((x    ), N)][mod((y + 1), N)] +
                    state->board[mod((x + 1), N)][mod((y + 1), N)];

                if (state->board[x][y] == ALIVE && (n_neigh == 2 || n_neigh == 3))
                    new_board[x][y] = ALIVE;
                else if (state->board[x][y] == DEAD && n_neigh == 3)
                    new_board[x][y] = ALIVE;
                else
                    new_board[x][y] = DEAD;
            }

        for (int x = 0; x < N; x++)
            for (int y = 0; y < N; y++)
                state->board[x][y] = new_board[x][y];
    }
}


////////////////////////////////////////////////////////////////////////////////!!
//SAND SIMULATION FUNCTIONS

//world_sim_puede_moverse define si un elemento se puede mover o interactura dada la presencia de otro elemento en su posicion destino
bool world_sim_puede_moverse(state_t *state, short sustancia, int x, int y){
    // Si las coordenadas se salen de los límites, no se puede mover por defecto
    if(x < 0 || x > N-1 || y < 0 || y > N-1){ return false;}
    switch (sustancia)
    {
    case SAND:
      if(state->board[x][y] == AIR){ return true;} 
      else if (state->board[x][y] == SAND){ return false;}
      else if (state->board[x][y] == WATER && drand48() < 0.5){ return true;}
      else if (state->board[x][y] == ROCK){ return false;}
      else if (state->board[x][y] == OIL){ return true;}
      else if (state->board[x][y] == HUMO){ return true;}
      
      break;

    case WATER:
      if(state->board[x][y] == AIR){ return true;} 
      else if (state->board[x][y] == SAND){ return false;}
      else if (state->board[x][y] == WATER){ return false;}
      else if (state->board[x][y] == ROCK){ return false;}
      else if (state->board[x][y] == OIL && drand48() < 0.5){ return true;}
      else if (state->board[x][y] == HUMO){ return true;}

      break;
  
    case ROCK:
      if(state->board[x][y] == AIR){ return true;} 
      else if (state->board[x][y] == SAND && drand48() < 0.2 ){ return true;}
      else if (state->board[x][y] == WATER && drand48() < 0.9 ){ return true;}
      else if (state->board[x][y] == ROCK){ return false;}
      else if (state->board[x][y] == OIL && drand48() < 0.9){ return true;}
      else if (state->board[x][y] == HUMO){ return true;}

      break;

    // No es necesario porque el aire nunca se mueve por si mismo
    /*case AIR: 
      if(state->board[x][y] == AIR){ return true;}
      break;*/

    case FIRE:
      if(state->board[x][y] == AIR){ return true;} 
      else if (state->board[x][y] == SAND){ return false;}
      else if (state->board[x][y] == WATER){ return true;}
      else if (state->board[x][y] == ROCK){ return false;}
      else if (state->board[x][y] == FIRE){ return false;}
      else if (state->board[x][y] == OIL){ return true;}
      else if (state->board[x][y] == HUMO){ return true;}
      break;

    case OIL:
      if(state->board[x][y] == AIR){ return true;} 
      else if (state->board[x][y] == SAND){ return false;}
      else if (state->board[x][y] == WATER){ return false;}
      else if (state->board[x][y] == ROCK){ return false;}
      else if (state->board[x][y] == OIL){ return false;}
      else if (state->board[x][y] == HUMO){ return true;}

      break;
    
    case HUMO:
      if(state->board[x][y] == AIR){ return true;} 
      
      break;
    
    default:
      return false;
      break;
    }

    return false;
}

//* all _sim_mover , define cuales seran las regls e interacciones que genera cada elemento al moverse

//La mejor forma seria importar la direccion a todo el arreglo de flags y trabajar con el, pero no pude hacerlo asi
void sand_sim_mover(state_t *state, bool seHaMovidoFlags[N][N], int fromX, int fromY, int toX, int toY){
    short sustancia = state->board[fromX][fromY];
    short otraSustancia = state->board[toX][toY];
    //Switch para las interacciones especiales
    switch (sustancia)
    {
    case FIRE:
        if(otraSustancia == OIL){
            state->board[fromX][fromY] =FIRE;
            state->board[toX][toY] = FIRE;
        } else if(otraSustancia == WATER){
            state->board[fromX][fromY] = HUMO;
            state->board[toX][toY] = WATER;
        } else{
            bool seDescompone= drand48() < 0.003;
            if(seDescompone){
                state->board[fromX][fromY] = HUMO;
                state->board[toX][toY] = otraSustancia;
            } else{
                state->board[fromX][fromY] = otraSustancia;
                state->board[toX][toY] = FIRE;
            }
  
        }
        break;

    case HUMO:
        
        if (true)
        {
            bool seDescompone= drand48() < 0.05;
            if(seDescompone){
            state->board[fromX][fromY] = AIR;
            state->board[toX][toY] = otraSustancia; } 
            else{
            state->board[fromX][fromY] = otraSustancia;
            state->board[toX][toY] = sustancia;
        }}

        break;

    default:
        state->board[fromX][fromY] = otraSustancia;
        state->board[toX][toY] = sustancia;
        break;
    }
    seHaMovidoFlags[fromX][fromY] = true;
    seHaMovidoFlags[toX][toY] = true;
}

bool sand_sim_mover_abajo(state_t *state, short sustancia, bool seHaMovidoFlags[N][N],int x, int y){
    if(world_sim_puede_moverse(state, sustancia, x, y + 1)){ //Mover abajo
        sand_sim_mover(state, seHaMovidoFlags, x, y, x, y + 1);
        return true;
    }
    return false;
}

bool sand_sim_mover_izq_der(state_t *state, short sustancia, bool seHaMovidoFlags[N][N],int x, int y){
    //random number to define if it should go left or right
    bool primeroIzquierda = drand48() < 0.5;
    if(primeroIzquierda){
        if(world_sim_puede_moverse(state, sustancia, x - 1, y)){ //Mover a la izquierda
            sand_sim_mover(state, seHaMovidoFlags, x, y, x - 1, y);
            return true;
        } else if(world_sim_puede_moverse(state, sustancia, x + 1, y)){ //Mover a la derecha
            sand_sim_mover(state, seHaMovidoFlags, x, y, x + 1, y);
            return true;
        }
    } else {
        if(world_sim_puede_moverse(state, sustancia, x + 1, y)){ //Mover a la derecha
            sand_sim_mover(state, seHaMovidoFlags, x, y, x + 1, y);
            return true;
        } else if(world_sim_puede_moverse(state, sustancia, x - 1, y)){ //Mover a la izquierda
            sand_sim_mover(state, seHaMovidoFlags, x, y, x - 1, y);
            return true;
        }
    }
    return false;
}

bool sand_sim_mover_abajo_diagonal(state_t *state, short sustancia, bool seHaMovidoFlags[N][N],int x, int y){
    //random number to define if it should go left or right
    bool primeroIzquierda = drand48() < 0.5;
        if(primeroIzquierda){
            if(world_sim_puede_moverse(state, sustancia, x - 1, y + 1)){ //Mover a la izquierda
                sand_sim_mover(state, seHaMovidoFlags, x, y, x-1, y+1);
                return true;
            } else if (world_sim_puede_moverse(state, sustancia, x + 1, y + 1)){ //Mover a la derecha
                sand_sim_mover(state, seHaMovidoFlags, x, y, x+1, y+1);
                return true;
            }
        } else {
            if(world_sim_puede_moverse(state, sustancia, x + 1, y + 1)){ //Mover a la derecha
                sand_sim_mover(state, seHaMovidoFlags, x, y, x+1, y+1);
                return true;
            } else if (world_sim_puede_moverse(state, sustancia, x - 1, y + 1)){ //Mover a la izquierda
                sand_sim_mover(state, seHaMovidoFlags, x, y, x-1, y+1);
                return true;
            }
        }
    return false;
}


// x, y: Posición de la partícula que se mueve abajo o a los lados
bool sand_sim_mover_abajo_y_lados(state_t *state, short sustancia, bool seHaMovidoFlags[N][N],int x, int y){
    
    //Si no se puede mover hacia abajo
    if(!sand_sim_mover_abajo(state, sustancia, seHaMovidoFlags, x, y)){                 
        //Se moverá en diagonal hacia abajo
        if(sand_sim_mover_abajo_diagonal(state, sustancia, seHaMovidoFlags, x, y)){
            //Si se mueve en diagonal, retornar true
            return true;
        }
    }
    return false; 
}


// x, y: Posición de la partícula que se mueve arriba o a los lados
bool sand_sim_mover_arriba_y_lados(state_t *state, short sustancia, bool seHaMovidoFlags[N][N],int x, int y){
    
    if(world_sim_puede_moverse(state, sustancia, x, y - 1)){ //Mover arriba
        sand_sim_mover(state, seHaMovidoFlags, x, y, x, y - 1);
        return true;
    }

    bool primeroIzquierda = drand48() < 0.5;

    if(primeroIzquierda){
        if(world_sim_puede_moverse(state, sustancia, x - 1, y - 1)){ //Mover a la izquierda
            sand_sim_mover(state, seHaMovidoFlags, x, y, x-1, y-1);
            return true;
        } else if (world_sim_puede_moverse(state, sustancia, x + 1, y - 1)){ //Mover a la derecha
            sand_sim_mover(state, seHaMovidoFlags, x, y, x+1, y-1);
            return true;
        }
    } else {
        if(world_sim_puede_moverse(state, sustancia, x + 1, y - 1)){ //Mover a la derecha
            sand_sim_mover(state, seHaMovidoFlags, x, y, x+1, y-1);
            return true;
        } else if (world_sim_puede_moverse(state, sustancia, x - 1, y - 1)){ //Mover a la izquierda
            sand_sim_mover(state, seHaMovidoFlags, x, y, x-1, y-1);
            return true;
        }
    }
    
    return false; 
}

//Variables benchmarking Función SandSim
int sandsim_framecnt = 0;
int thread_sandsim_cnt = 1;
int fps_sandsim = 0;
//double time_sandsim_acum = 0;
double avg_time_sandsim = 0;
double avg_FPS_sandsim = 0;


//* /////////////////////////////////
//* cuda function
__global__ void cudaFunction(int *d_Matrix, int *d_Values, int dim) {

  // map from threadIdx/BlockIdx to pixel position
  int x = threadIdx.x + blockIdx.x * blockDim.x;// thread column
  int y = threadIdx.y + blockIdx.y * blockDim.y;//thread row

  *d_Matrix = {{1}};

  if (x<dim && y<dim) {
    int offset = x + y * blockDim.x * gridDim.x;
    // write array contINING all the offset variable values
    *d_Values[offset] = offset;
  }
  
  
}
//* /////////////////////////////////


//***** world_sand_sim() RUNS THE SIMULATION logic for all elements of the world
void world_sand_sim(SDL_Renderer *renderer, state_t *state)
{

  //*/////////////////////////////////////
  //CUDA CODE

  int grid_dim;
  int block_dim;

  if (N < 1024) {
      grid_dim = N;
      block_dim = N;
  }
  else {
      grid_dim = 1024;
      block_dim = 1024;
  }

  // malloc device memory
  int Matrix[N][N]={{0}};
  int Values[N*N]={{0}};
  int *d_Matrix;
  int *d_Values;

  HANDLE_ERROR( hipMalloc((void **)&d_Matrix, N * N * sizeof(int)));
  HANDLE_ERROR( hipMalloc((void **)&d_Values, N * N * sizeof(int)));

  // transfer data from host to device
  //HANDLE_ERROR( hipMemcpy(d_Matrix, &Matrix, hipMemcpyHostToDevice));

  // invoke the kernel
  cudaFunction<<< grid_dim, block_dim >>>(d_Matrix, d_Values, N);
  hipDeviceSynchronize();


  HANDLE_ERROR( hipMemcpy( &Matrix, d_Matrix, N * N * sizeof(int) , hipMemcpyDeviceToHost ) );
  HANDLE_ERROR( hipMemcpy( &Values, d_Values, N * N * sizeof(int) , hipMemcpyDeviceToHost ) );

  printf( Matrix );
  printf( Values );

  // free host and devide memory
  HANDLE_ERROR( hipFree(d_MatA));
  free(h_A);


  //*/////////////////////////////////////



//   ++sandsim_framecnt; //Sumando los fps de sandsim

//   if (state->mode == RUNNING_MODE){

//       //CÁLCULO DE EL TIEMPO PROMEDIO PARA CADA NUMERO DE THREADS
//       if(threads != thread_sandsim_cnt){
//           double avg_fps = sandsim_framecnt/s_cambio_threads;
//           long double avg_time = (avg_time_sandsim /avg_fps) / s_cambio_threads; 
//           SDL_Log("[SANDSIM] Tiempo promedio para %d threads (ms): %0.1Lf",thread_sandsim_cnt,avg_time);
//           avg_time_sandsim = 0; //Reinciar el conteo del promedio acumulado en X segundos
//           sandsim_framecnt = 0;
//           thread_sandsim_cnt = threads; //Se actualiza la variable para el contador interno de sandsim
//       }

      
//     for (int i = 0; i < MOVES_PER_FRAME; i++) {
//       //int new_board[N][N] = {state->board};
//       bool seHaMovidoFlags[N][N] = {false};

//       //*calculate time to render the grid
//       struct timeval tval_before_sandsim, tval_after_sandsim, tval_result_sandsim;
//       gettimeofday(&tval_before_sandsim, NULL);
      
      
//       //#pragma omp parallel num_threads(threads) 
//       //{
        
//         for (int y = N-1; y >= 0; y--){
//             //#pragma omp for //collapse(2)
//             for (int x = 0; x < N; x++) {
                
//                 //Saltar si ya se ha movido esta posicion
//                 if(seHaMovidoFlags[x][y]) continue;
                
//                 //*g  rules and functions for sand
//                 if(state->board[x][y] == SAND){
//                     sand_sim_mover_abajo_y_lados(state, SAND, seHaMovidoFlags, x, y);                  
//                 }

//                 //*g rules and functions for rock
//                 if(state->board[x][y] == ROCK){
//                     sand_sim_mover_abajo(state, ROCK, seHaMovidoFlags, x, y);
//                 }

//               //*g  rules and functions for water
//                 if(state->board[x][y] == WATER){
                    
//                     //Si el agua no se puede mover abajo o a los lados
//                     if(!sand_sim_mover_abajo_y_lados(state, WATER, seHaMovidoFlags, x, y)){
//                         //Se mueve a la izquierda o derecha
//                         sand_sim_mover_izq_der(state, WATER, seHaMovidoFlags, x, y);
//                     }   
//                 }

//               //*g  rules and functions for oil
//                 if(state->board[x][y] == OIL){
//                     //Si el agua no se puede mover abajo o a los lados
//                     if(!sand_sim_mover_abajo_y_lados(state, OIL, seHaMovidoFlags, x, y)){
//                         //Se mueve a la izquierda o derecha
//                         sand_sim_mover_izq_der(state, OIL, seHaMovidoFlags, x, y);
//                     }   
//                 }

//                 //*g rules and functions for fire
//                 if(state->board[x][y] == FIRE){
//                     if (!sand_sim_mover_abajo_y_lados(state, FIRE, seHaMovidoFlags, x, y))
//                     {
//                       bool seDescompone= drand48() < 0.2;
//                       if(seDescompone){
//                           state->board[x][y] = HUMO;
//                       } 

//                     }
                    
//                 }

//                 //*g rules and functions for humo
//                 if(state->board[x][y] == HUMO){
//                   //Si el humo no puede moverse arriba o diagonal va para los lados
//                     if(!sand_sim_mover_arriba_y_lados(state, HUMO, seHaMovidoFlags, x, y)){
//                         //Se mueve a la izquierda o derecha
//                         sand_sim_mover_izq_der(state, HUMO, seHaMovidoFlags, x, y);
//                     }   
                    
//                 }

//             }
//         }  

          
//         //*calculate time to render the grid
//         gettimeofday(&tval_after_sandsim, NULL);

//         timersub(&tval_after_sandsim, &tval_before_sandsim, &tval_result_sandsim);


//       //  #pragma omp single nowait
//        // {
//           //Calculo FPS
//           //Si ha pasado un segundo desde la ultima medicion
//         /*  if((tval_after_sandsim.tv_sec - getTimerS(TVAL_SANDSIM)) != 0){
//               fps_sandsim = fps_sandsim_cnt; //Capturar cuantas veces se ha ejecutado esta funcion (FPS)
//               fps_sandsim_cnt = 0; //Reiniciar la cuenta de FPS
//               avg_time_sandsim += time_sandsim_acum / (long int)fps_sandsim; //Tiempos de cada frame/ FPS = prom. tiempo ejecución por segundo
//               //SDL_Log("%lf",avg_time_sandsim);
//               time_sandsim_acum = 0; //Reiniciar el acumulador del tiempo
//               resetTimer(TVAL_SANDSIM); //Actualizar timer

//           } else{  //Si no ha pasado el segundo
//               ++fps_sandsim_cnt; //Ir sumando los frames
//               time_sandsim_acum += tval_result_sandsim.tv_usec; //Va acumulando todos los tiempos de cada frame
//           }
//         */

//           avg_time_sandsim += tval_result_sandsim.tv_usec;

//         char str[128];
//         sprintf(str, "Total time to execute function world_sand_sim (ms): %ld", 
//             (long int)tval_result_sandsim.tv_usec);
//         renderFormattedText(renderer, str, 0 , 40);
//         }
//       //}

//   }
//  // }  
    
}
